#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>

int main() {
  int device_count;
  hipGetDeviceCount(&device_count);
  std::cout<<"Device count: "<<device_count<<std::endl;
  hipDeviceProp_t device_prop;
  hipGetDeviceProperties(&device_prop, 0);
  std::cout<<"Max threads per block: "<<device_prop.maxThreadsPerBlock<<std::endl;
  std::cout<<"Multiprocessor count: "<<device_prop.multiProcessorCount<<std::endl;
  std::cout<<"Device clock rate: "<<device_prop.clockRate<<std::endl;
  return 0;
}
